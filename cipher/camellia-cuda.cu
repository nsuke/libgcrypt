
#include <hip/hip_runtime.h>
#define CAMELLIA_BLOCK_SIZE 16
#define CAMELLIA_TABLE_BYTE_LEN 272
#define CAMELLIA_TABLE_WORD_LEN (CAMELLIA_TABLE_BYTE_LEN / 4)

typedef unsigned int u64;

typedef unsigned int KEY_TABLE_TYPE[CAMELLIA_TABLE_WORD_LEN];

typedef struct {
  KEY_TABLE_TYPE keytable;
  int keybitlength;
#ifdef USE_AESNI_AVX
  unsigned int
      use_aesni_avx : 1; /* AES-NI/AVX implementation shall be used.  */
#endif                   /*USE_AESNI_AVX*/
#ifdef USE_AESNI_AVX2
  unsigned int
      use_aesni_avx2 : 1; /* AES-NI/AVX2 implementation shall be used.  */
#endif                    /*USE_AESNI_AVX2*/
} CAMELLIA_context;

extern "C" {
void _gcry_camellia_cuda_ocb_enc(CAMELLIA_context* ctx,
                                 unsigned char* out,
                                 const unsigned char* in,
                                 unsigned char* offset,
                                 unsigned char* checksum,
                                 const u64 Ls[32]);

void _gcry_camellia_cuda_ocb_dec(CAMELLIA_context* ctx,
                                 unsigned char* out,
                                 const unsigned char* in,
                                 unsigned char* offset,
                                 unsigned char* checksum,
                                 const u64 Ls[32]);
}

__global__ void VecAdd(float* A, float* B, float* C) {
  int i = threadIdx.x;
  C[i] = A[i] + B[i];
}

void _gcry_camellia_cuda_ocb_enc(CAMELLIA_context* ctx,
                                 unsigned char* out,
                                 const unsigned char* in,
                                 unsigned char* offset,
                                 unsigned char* checksum,
                                 const u64 Ls[32]) {}

void _gcry_camellia_cuda_ocb_dec(CAMELLIA_context* ctx,
                                 unsigned char* out,
                                 const unsigned char* in,
                                 unsigned char* offset,
                                 unsigned char* checksum,
                                 const u64 Ls[32]) {}
